/*
The data set that we are using has 4 attributes but we are using only 2 attributes.
Those 2 attributes are 1)Study Time 2) Exam Performance.
These 2 attributes will be used to calculate the student's "KnowledgeLevel"
KnowledgeLevel can be High or Low in our program but in the data set "KnowledgeLevel" has High, Low and Middle.
We will represent High as 1 and low as 0. For now we will consider Middle as High so it will be 1.
*/
#include<stdio.h>
#include<math.h>
#include<string.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
void readDataset(); /*<--- This function reads data set into below arrays.*/
void displayDataset();/*<--- This function will display our data set*/
__global__ void startClustering(float*,float*,float*,int,int,int,int,int,int,int,int,int,int,struct cluster*, struct cluster*);
void remapSeedValues();
void displayClusters();
int highCluster = 1, lowCluster = 1;
float study_time[260]/*<--- attribute number: 1*/, examPerformance[260]/*<--- attribute number: 2*/, targetAnswers[260]/*<---- real knowledge level: 3*/;
float ourAnswer[258]/*<--- this will store our answer for knowledge level 1 = High, 0 = Low*/;

/*sample seed value to create 2 clusters or knowledge levels*/
/*These seed values will change during different iterations*/
float svLX = 0.2, svLY = 0.2, svHX = 0.78, svHY = 0.78;
float psvLX, psvLY, psvHX, psvHY;
float seedDistance = 0.0;
/*examPerformance is on Y-axis and study_time is on X-axis*/

struct cluster
{
	float study_time;
	float exam_performance;
	float target_value;
}cluster_one[260], cluster_two[250];
int cluster_one_index = 0, cluster_two_index = 0;
__global__ void startClustering(float *study_time, float *examPerformance, float *targetAnswers,int N, int svHX, int svHY, int svLX, int svLY, int seedDistance, int lowCluster, int highCluster, int cluster_one_index, int cluster_two_index, struct cluster *cluster_one, struct cluster *cluster_two)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float distance_1, distance_2;
	
	distance_1 = sqrt(((svHX-study_time[idx])*(svHX-study_time[idx])) + ((svHY-examPerformance[idx])*(svHY-examPerformance[idx])));
	distance_2 = sqrt(((svLX-study_time[idx])*(svLX-study_time[idx])) + ((svLY-examPerformance[idx])*(svLY-examPerformance[idx])));
	
	if((distance_1 <= seedDistance) && (highCluster==1))
	{
		cluster_one[cluster_one_index].study_time = study_time[idx];
		cluster_one[cluster_one_index].exam_performance = examPerformance[idx];
		cluster_one[cluster_one_index++].target_value = targetAnswers[idx];
	}
	else if((distance_2 <= seedDistance) && (lowCluster==1))
	{
		cluster_two[cluster_two_index].study_time = study_time[idx];
		cluster_two[cluster_two_index].exam_performance = examPerformance[idx];
		cluster_two[cluster_two_index++].target_value = targetAnswers[idx];
	}
}
float calculateDistanceSerial(float x1, float x2, float y1,float y2)
{
	return sqrt(((x2-x1)*(x2-x1))+((y2-y1)*(y2-y1)));
}
void startParallelProcessing()
{
	float *sT, *eP, *tA;
	struct cluster *c1, *c2;
	int N = 260;
	size_t size = N * sizeof(float);
	hipMalloc((void **) &sT, size);
	hipMalloc((void **) &eP, size);
	hipMalloc((void **) &tA, size);
	hipMemcpy(sT, study_time, size, hipMemcpyHostToDevice);
	hipMemcpy(eP, examPerformance, size, hipMemcpyHostToDevice);
	hipMemcpy(tA, targetAnswers, size, hipMemcpyHostToDevice);	
	size = N * sizeof(struct cluster);
	hipMalloc((void **) &c1, size);
	hipMalloc((void **) &c2, size);
	hipMemcpy(c1, cluster_one, size, hipMemcpyHostToDevice);
	hipMemcpy(c2, cluster_two, size, hipMemcpyHostToDevice);
	int block_size = 1;
  	int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
	startClustering <<< n_blocks, block_size >>> (sT, eP, tA, N, svHX, svHY, svLX, svLY, seedDistance, lowCluster, highCluster, cluster_one_index, cluster_two_index, c1, c2);
	remapSeedValues();
	if(psvHX == svHX && psvHY == svHY)
	{
		printf("\nHigh Knowledge Cluster Seed Value Achieved Successfully.\n");
		highCluster = 0;
	}
	if(psvLX = svLX && psvLY == svLY)
	{
		printf("\nLow Knowledge Cluster Seed Value Achieved Successfully.\n");
		lowCluster = 0;
	}
	if(lowCluster==1 || highCluster==1)
	{
		startClustering <<< n_blocks, block_size >>> (sT, eP, tA, N, svHX, svHY, svLX, svLY, seedDistance, lowCluster, highCluster, cluster_one_index, cluster_two_index, c1, c2);
	}
	hipMemcpy(cluster_one, c1, sizeof(struct cluster)*N, hipMemcpyDeviceToHost);
	hipMemcpy(cluster_two, c2, sizeof(struct cluster)*N, hipMemcpyDeviceToHost);
	hipFree(sT);
	hipFree(eP);
	hipFree(tA);
	hipFree(c1);
	hipFree(c2);
}
int main()
{
	//freopen("output.txt","w",stdout);
	readDataset();
	psvLX = svLX; psvLY = svLY; psvHX = svHX; psvHY = svHY;
	seedDistance = calculateDistanceSerial(svLX,svHX,svLY,svHY);
	displayDataset();
	startParallelProcessing();
	displayClusters();
	return 0;
}
void remapSeedValues()
{
	int counter, count = 0;
	float StudyTime = 0, ExamPerformance = 0;
	psvHX = svHX;
	psvHY = svHY;
	psvLX = svLX;
	psvLY = svLY;
	for(counter=2;counter<260;counter++)
	{
		if(cluster_one[counter].study_time==0 && cluster_one[counter].exam_performance==0)
		{
			break;
		}
		count++;
		StudyTime = cluster_one[counter].study_time + StudyTime;
		ExamPerformance = cluster_one[counter].exam_performance + ExamPerformance;
	}
	svHX = StudyTime/count;
	svHY = ExamPerformance/count;
	ExamPerformance = StudyTime = 0;
	count = 0;
	for(counter=2;counter<260;counter++)
	{
		if(cluster_two[counter].study_time==0 && cluster_two[counter].exam_performance==0)
		{
			break;
		}
		count++;
		StudyTime = cluster_one[counter].study_time + StudyTime;
		ExamPerformance = cluster_one[counter].exam_performance + ExamPerformance;
	}
	svLX = StudyTime/count;
	svLY = ExamPerformance/count;
	printf("\nPrevious Low X: %f\tNew Low X: %f\nPrevious Low Y: %f\tNew Low Y: %f\n",psvLX,svLX,psvLY,svLY);
	printf("\nPrevious High X: %f\tNew High X: %f\nPrevious High Y: %f\tNew High Y: %f\n",psvHX,svHX,psvHY,svHY);
	seedDistance = calculateDistanceSerial(svLX,svHX,svLY,svHY);
	/*count = 0;
	for(counter=2;counter<260;counter++)
	{
		if(cluster_three[counter].study_time==0 && cluster_three[counter].examPerformance==0)
		{
			break;
		}
		count++;
		StudyTime = cluster_one[counter].study_time + StudyTime;
		ExamPerformance = cluster_one[counter].examPerformance + ExamPerformance;
	}*/
}
void readDataset()
{
	FILE *ptr_file;
    char buf[1000];
	ptr_file =fopen("input.txt","r");
    if (!ptr_file)
	{
		printf("Requested Input File Not Found :(");
		return;
	}
	
	int studyTimeIndex = 0, examPerformanceIndex = 0, targetAnswersIndex = 0;
	
	while (fgets(buf,1000, ptr_file)!=NULL)
	{
		int counter;
		char *p = strtok(buf,"\t");
		for(counter = 0;counter<=5 && p!=NULL;counter++)
		{
			if(p!=NULL)
			{
				switch(counter)
				{
					case 0:
						study_time[studyTimeIndex++] = atof(p);
						break;
					case 4:
						examPerformance[examPerformanceIndex++] = atof(p);
						break;
					case 5:
						if(strcmp(p,"High")==0 || strcmp(p,"Middle")==0)
						{
							targetAnswers[targetAnswersIndex++] = 1;
						}
						else
						{
							targetAnswers[targetAnswersIndex++] = 0;
						}
						break;
					default:
						break;
				}
			}
			p = strtok(NULL,"\t");
		}
	}
	fclose(ptr_file);
}
void displayDataset()
{
	printf("\n\nDisplaying Dataset Entries\n\nStudy Time\tExam Performance\tKnowledge Level\n\n");
	int counter = 0;
	for(counter = 0;counter<259;counter++)
	{
		printf("%f\t%f\t\t%f\n",study_time[counter],examPerformance[counter],targetAnswers[counter]);
	}
	printf("\n\nEnd of Displaying Dataset\n\n");
}
void displayClusters()
{
	int counter, total_ones = 0, total_zeroes = 0, total = 0;
	printf("\n\nDisplaying Cluster of Users with High KnowledgeLevel: \nStudy Time\tExam Performance\tTarget Cluster (1: High, 0: Low)\n\n");
	for(counter = 2;counter<260;counter++)
	{
		if(cluster_one[counter].study_time==0 && cluster_one[counter].exam_performance==0)
		{
			break;
		}
		if(cluster_one[counter].target_value==1.0)
		{
			total_ones++;
		}
		if(cluster_one[counter].target_value==0.0)
		{
			total_zeroes++;
		}
		total++;
		printf("%f\t%f\t\t%f\n",cluster_one[counter].study_time, cluster_one[counter].exam_performance, cluster_one[counter].target_value);
	}
	printf("\n\nTotal Entries: %d\tLow Entried: %d\tHigh Entries: %d\n",total,total_zeroes,total_ones);
	printf("-------------------------------------------------------------------------------");
	printf("\n\nDisplaying Cluster of Users with Low KnowledgeLevel: \nStudy Time\tExam Performance\tTarget Cluster (1: High, 0: Low)\n\n");
	total = total_zeroes = total_ones = 0;	
	for(counter = 0;counter<260;counter++)
	{
		if(cluster_two[counter].study_time==0 && cluster_two[counter].exam_performance==0)
		{
			break;
		}
		if(cluster_two[counter].target_value==1.0)
		{
			total_ones++;
		}
		if(cluster_two[counter].target_value==0.0)
		{
			total_zeroes++;
		}
		total++;
		printf("%f\t%f\t\t%f\n",cluster_two[counter].study_time, cluster_two[counter].exam_performance, cluster_two[counter].target_value);
	}
	printf("\n\nTotal Entries: %d\tLow Entried: %d\tHigh Entries: %d\n",total,total_zeroes,total_ones);
	printf("--------------------------------------------------------------------------------");
}
